#include "hip/hip_runtime.h"
#include <stdio.h>

extern "C" void THError(const char *fmt, ...);
extern "C" int cuda_maphostmem;

#define errcheck(f) do {int rc = f; if(rc) THError("Error %d in line %s:%d", rc, __FILE__, __LINE__); } while(0)

#define BYTE2FLOAT 0.003921568f // 1/255

__global__ void rgb2float_kernel(float *dst, const unsigned char *src, int width, int height, int srcstride, const float *mean, const float *std)
{
	int c;

	for(c = 0; c < 3; c++)
	{
		dst[4*threadIdx.x + (blockIdx.x + c * height) * width] =
			(src[c + 3*4*threadIdx.x + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c];
		dst[4*threadIdx.x+1 + (blockIdx.x + c * height) * width] =
			(src[c + 3*(4*threadIdx.x+1) + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c];
		dst[4*threadIdx.x+2 + (blockIdx.x + c * height) * width] =
			(src[c + 3*(4*threadIdx.x+2) + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c];
		dst[4*threadIdx.x+3 + (blockIdx.x + c * height) * width] =
			(src[c + 3*(4*threadIdx.x+3) + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c];
	}
}

__global__ void rgb2half_kernel(unsigned short *dst, const unsigned char *src, int width, int height, int srcstride, const float *mean, const float *std)
{
	int c;

	for(c = 0; c < 3; c++)
	{
		dst[4*threadIdx.x + (blockIdx.x + c * height) * width] =
			__float2half_rn((src[c + 3*4*threadIdx.x + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c]);
		dst[4*threadIdx.x+1 + (blockIdx.x + c * height) * width] =
			__float2half_rn((src[c + 3*(4*threadIdx.x+1) + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c]);
		dst[4*threadIdx.x+2 + (blockIdx.x + c * height) * width] =
			__float2half_rn((src[c + 3*(4*threadIdx.x+2) + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c]);
		dst[4*threadIdx.x+3 + (blockIdx.x + c * height) * width] =
			__float2half_rn((src[c + 3*(4*threadIdx.x+3) + srcstride*blockIdx.x] * BYTE2FLOAT - mean[c]) / std[c]);
	}
}

extern "C" float *cuda_rgb2float(float *dst, const unsigned char *src, int width, int height, int srcstride, const float *mean, const float *std);
extern "C" float *cuda_rgb2half(float *dst, const unsigned char *src, int width, int height, int srcstride, const float *mean, const float *std);

float *cuda_rgb2float(float *dst, const unsigned char *src, int width, int height, int srcstride, const float *mean, const float *std)
{
	unsigned char *csrc;
	float *cmean, *cstd;
	
	if(cuda_maphostmem)
	{
		if(cuda_maphostmem == 2)
			errcheck(hipHostRegister((void *)src, height*srcstride, hipHostRegisterMapped));
		errcheck(hipHostGetDevicePointer((void **)&csrc, (void *)src, 0));
	} else {
		errcheck(hipMalloc((void **)&csrc, height * srcstride));
		errcheck(hipMemcpy(csrc, src, height * srcstride, hipMemcpyHostToDevice));
	}
	errcheck(hipMalloc((void **)&cmean, 3 * sizeof(*cmean)));
	errcheck(hipMemcpy(cmean, mean, 3 * sizeof(*cmean), hipMemcpyHostToDevice));
	errcheck(hipMalloc((void **)&cstd, 3 * sizeof(*cstd)));
	errcheck(hipMemcpy(cstd, std, 3 * sizeof(*std), hipMemcpyHostToDevice));

	rgb2float_kernel<<<height, width/4>>>(dst, csrc, width, height, srcstride, cmean, cstd);
	errcheck(hipDeviceSynchronize());
	
	if(cuda_maphostmem == 2)
		hipHostUnregister((void *)src);
	else if(cuda_maphostmem == 0)
		hipFree(csrc);
	hipFree(cmean);
	hipFree(cstd);
	
	return dst;
}

float *cuda_rgb2half(float *dst, const unsigned char *src, int width, int height, int srcstride, const float *mean, const float *std)
{
	unsigned char *csrc;
	float *cmean, *cstd;
	
	if(cuda_maphostmem)
	{
		if(cuda_maphostmem == 2)
			errcheck(hipHostRegister((void *)src, height*srcstride, hipHostRegisterMapped));
		errcheck(hipHostGetDevicePointer((void **)&csrc, (void *)src, 0));
	} else {
		errcheck(hipMalloc((void **)&csrc, height * srcstride));
		errcheck(hipMemcpy(csrc, src, height * srcstride, hipMemcpyHostToDevice));
	}
	errcheck(hipMalloc((void **)&cmean, 3 * sizeof(*cmean)));
	errcheck(hipMemcpy(cmean, mean, 3 * sizeof(*cmean), hipMemcpyHostToDevice));
	errcheck(hipMalloc((void **)&cstd, 3 * sizeof(*cstd)));
	errcheck(hipMemcpy(cstd, std, 3 * sizeof(*std), hipMemcpyHostToDevice));

	rgb2half_kernel<<<height, width/4>>>((unsigned short *)dst, csrc, width, height, srcstride, cmean, cstd);
	errcheck(hipDeviceSynchronize());
	
	if(cuda_maphostmem)
		hipHostUnregister((void *)src);
	else hipFree(csrc);
	hipFree(cmean);
	hipFree(cstd);
	
	return dst;
}
